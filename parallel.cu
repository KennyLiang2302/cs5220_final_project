#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/gather.h>
#include <thrust/adjacent_difference.h>
#include <thrust/count.h>
#include <thrust/tuple.h>
#include <thrust/logical.h>
#include <vector>
#include "common.h"
#include <random>
#include <thrust/random.h>
#include <omp.h>
#include <algorithm>
#include <thrust/execution_policy.h>
#include <thrust/partition.h>

// GLOBAL VARIABLES
thrust::device_vector<double> *x_train_global;
thrust::device_vector<double> *y_train_global;
std::vector<double> x_test_global;
std::vector<double> y_test_global;
int train_size_global;
int test_size_global;
int D_global;

std::vector<double> predictions_global;

// HELPER FUNCTIONS

__device__ int round_down(int x, int D)
{
    return x - (x % D);
}

// TODO, can try to optimize this function.
template <typename DerivedPolicy>
thrust::device_vector<int> idif_gpu(const thrust::detail::execution_policy_base<DerivedPolicy> &exec_policy, thrust::device_vector<double> &x, int N)
{
    // Find differences between adjacent values and store in diffs vector
    thrust::device_vector<double> adj_diffs(N);
    thrust::adjacent_difference(exec_policy, x.begin(), x.end(), adj_diffs.begin());

    // Make a zipped iterator that iterates diffs and idx
    thrust::device_vector<int> idx(N);
    thrust::sequence(exec_policy, idx.begin(), idx.end());
    thrust::zip_iterator<thrust::tuple<thrust::device_vector<int>::iterator,
                                       thrust::device_vector<double>::iterator>>
        zipped_iterator = thrust::make_zip_iterator(thrust::make_tuple(idx.begin(), adj_diffs.begin()));

    // Count_if the number of elements that are different w.r.t. threshold
    int size = thrust::count_if(exec_policy, zipped_iterator, zipped_iterator + idx.size(), [] __device__(thrust::tuple<int, double> pair)
                                { return thrust::get<1>(pair) > THRESHOLD && thrust::get<0>(pair) != 0; });
    thrust::device_vector<thrust::tuple<int, double>> idif_tuples(size);

    // Use copy_if to store indices
    thrust::copy_if(exec_policy,
                    zipped_iterator, zipped_iterator + idx.size(), idif_tuples.begin(), [] __device__(thrust::tuple<int, double> pair)
                    { return thrust::get<1>(pair) > THRESHOLD && thrust::get<0>(pair) != 0; });

    // Use transform to subtract 1 from indices
    thrust::device_vector<int> idif_idx(size);
    thrust::transform(exec_policy, idif_tuples.begin(), idif_tuples.end(), idif_idx.begin(), [] __device__(thrust::tuple<int, double> pair)
                      { return thrust::get<0>(pair) - 1; });

    return idif_idx;
}

template <typename DerivedPolicy>
thrust::device_vector<int> argsort_gpu(const thrust::detail::execution_policy_base<DerivedPolicy> &exec_policy, thrust::device_vector<double> &x, int d, int D, int N)
{
    thrust::device_vector<int> indices(N);
    thrust::sequence(exec_policy, indices.begin(), indices.end());

    double *x_ptr = thrust::raw_pointer_cast(x.data());

    thrust::stable_sort(exec_policy, indices.begin(), indices.end(), [x_ptr, D, d] __device__(int left_idx, int right_idx)
                        { return x_ptr[left_idx * D + d] < x_ptr[right_idx * D + d]; });
    return indices;
}

template <typename DerivedPolicy>
split_output_t split_gpu(const thrust::detail::execution_policy_base<DerivedPolicy> &exec_policy, int D, int N, thrust::device_vector<double> &x_train, thrust::device_vector<double> &y_train)
{
    double weight = 1.0 / N;

    thrust::device_vector<split_output_t> split_results((N - 1) * D);

    split_output_t init_output;
    init_output.cut_feature = std::numeric_limits<int>::infinity();
    init_output.cut_value = std::numeric_limits<double>::infinity();
    init_output.loss = std::numeric_limits<double>::infinity();
    thrust::fill(exec_policy, split_results.begin(), split_results.end(), init_output);

    // iterate through each feature
    for (int d = 0; d < D; ++d)
    {
        thrust::device_vector<int> indices_y = argsort_gpu(exec_policy, x_train, d, D, N);

        thrust::device_vector<int> indices_x(N);
        thrust::transform(exec_policy, indices_y.begin(), indices_y.end(), indices_x.begin(),
                          [D, d] __device__(int y_idx) -> int
                          { return y_idx * D + d; });

        thrust::device_vector<double> x_train_sorted(N);
        thrust::device_vector<double> y_train_sorted(N);

        thrust::gather(exec_policy, indices_x.begin(), indices_x.end(), x_train.begin(), x_train_sorted.begin());
        thrust::gather(exec_policy, indices_y.begin(), indices_y.end(), y_train.begin(), y_train_sorted.begin());

        thrust::device_vector<double> y_train_sorted_squared(N);
        thrust::transform(exec_policy, y_train_sorted.begin(), y_train_sorted.end(), y_train_sorted_squared.begin(),
                          [] __device__(double x) -> double
                          { return x * x; });

        double mean_square_right = weight * thrust::reduce(exec_policy, y_train_sorted_squared.begin(), y_train_sorted_squared.end(), 0.0);
        double mean_right = weight * thrust::reduce(exec_policy, y_train_sorted.begin(), y_train_sorted.end(), 0.0);

        thrust::device_vector<double> y_prefix_sum(N);
        thrust::device_vector<double> y_squared_prefix_sum(N);

        thrust::inclusive_scan(exec_policy, y_train_sorted.begin(), y_train_sorted.end(), y_prefix_sum.begin());
        thrust::inclusive_scan(exec_policy, y_train_sorted_squared.begin(), y_train_sorted_squared.end(), y_squared_prefix_sum.begin());

        thrust::device_vector<int> idif_indices = idif_gpu(exec_policy, x_train_sorted, N);

        split_output_t *split_results_ptr = thrust::raw_pointer_cast(split_results.data());
        double *x_train_sorted_ptr = thrust::raw_pointer_cast(x_train_sorted.data());

        double *y_prefix_sum_ptr = thrust::raw_pointer_cast(y_prefix_sum.data());
        double *y_squared_prefix_sum_ptr = thrust::raw_pointer_cast(y_squared_prefix_sum.data());

        // todo go through idif indices
        thrust::for_each(exec_policy, idif_indices.begin(), idif_indices.end(),
                         [N, D, d, weight, mean_square_right, mean_right, split_results_ptr, y_prefix_sum_ptr, y_squared_prefix_sum_ptr, x_train_sorted_ptr] __device__(int index)
                         {
                             double mean_square_left = weight * y_squared_prefix_sum_ptr[index];
                             double mean_left = weight * y_prefix_sum_ptr[index];
                             double local_mean_right = mean_right - mean_left;
                             double local_mean_sq_right = mean_square_right - mean_square_left;
                             double weight_left = (index + 1) * weight;
                             double weight_right = (N - index - 1) * weight;
                             double left_loss = mean_square_left - (mean_left * mean_left) / weight_left;
                             double right_loss = local_mean_sq_right - (local_mean_right * local_mean_right) / weight_right;
                             split_output_t split_output;
                             split_output.cut_feature = d;
                             split_output.cut_value = (x_train_sorted_ptr[index] + x_train_sorted_ptr[index + 1]) / 2;
                             split_output.loss = left_loss + right_loss;
                             split_results_ptr[(N - 1) * d + index] = split_output;
                         });

        hipDeviceSynchronize();
    }

    split_output_t output = thrust::reduce(exec_policy, split_results.begin(), split_results.end(), init_output,
                                           [] __device__ __host__(split_output_t left, split_output_t right)
                                           {
                                               if (left.loss < right.loss)
                                               {
                                                   return left;
                                               }
                                               else
                                               {
                                                   return right;
                                               }
                                           });

    return output;
}

/** Checks that all elements in a vector are equal to a value within some error
 */
template <typename DerivedPolicy>
bool elements_equal_gpu(const thrust::detail::execution_policy_base<DerivedPolicy> &exec_policy, thrust::device_vector<double> &values, int size,
                        double epsilon)
{
    thrust::device_ptr<double> values_ptr = values.data();
    return thrust::count_if(exec_policy, values.begin(), values.end(),
                            [=] __device__(double x)
                            {
                                return fabs(x - values_ptr[0]) > epsilon;
                            }) == 0;
}

/** Checks that all rows are equal within some error*/
template <typename DerivedPolicy>
bool rows_equal_gpu(const thrust::detail::execution_policy_base<DerivedPolicy> &exec_policy, thrust::device_vector<double> &d_x, int D, int N, double epsilon)
{
    // TODO, same as above
    thrust::device_ptr<double> d_x_ptr = d_x.data();

    return thrust::count_if(exec_policy, thrust::make_counting_iterator(0), thrust::make_counting_iterator(N - 1),
                            [=] __device__(int i)
                            {
                                for (int j = 0; j < D; ++j)
                                {
                                    if (fabs(d_x_ptr[i * D + j] - d_x_ptr[(i + 1) * D + j]) > epsilon)
                                    {
                                        return true;
                                    }
                                }
                                return false;
                            }) == 0;
}

tree_node_t *build_cart_helper(int D, int N, thrust::device_vector<double> &d_x_train, thrust::device_vector<double> &d_y_train, int depth)
{
    double weight = 1.0 / N;
    double mean = thrust::reduce(thrust::device, d_y_train.begin(), d_y_train.end(), 0.0) / N;

    // if no more branching can be done, return a leaf node
    if (depth == 0 || elements_equal_gpu(thrust::device, d_y_train, N, THRESHOLD) || rows_equal_gpu(thrust::device, d_x_train, D, N, THRESHOLD))
    {
        tree_node_t *leaf = (tree_node_t *)malloc(sizeof(tree_node_t));
        leaf->left = NULL;
        leaf->right = NULL;
        leaf->parent = NULL;
        leaf->prediction = mean;
        leaf->cut_feature = -1;
        leaf->cut_value = NAN;
        return leaf;
    }
    else
    {
        thrust::device_ptr<double> d_x_train_ptr = d_x_train.data();

        split_output_t split = split_gpu(thrust::device, D, N, d_x_train, d_y_train);

        int size_left = thrust::count_if(thrust::device,
                                         thrust::make_counting_iterator(0), thrust::make_counting_iterator(N),
                                         [d_x_train_ptr, split, D] __device__(int i)
                                         {
                                             double x = d_x_train_ptr[i * D + split.cut_feature];
                                             return x <= split.cut_value;
                                         });
        int size_right = N - size_left;

        thrust::device_vector<int> left_indices(size_left);
        thrust::device_vector<int> right_indices(size_right);
        thrust::device_vector<int> left_x_indices(size_left * D);
        thrust::device_vector<int> right_x_indices(size_right * D);

        thrust::stable_partition_copy(thrust::device,
                                      thrust::make_counting_iterator(0), thrust::make_counting_iterator(N),
                                      left_indices.begin(), right_indices.begin(),
                                      [=] __device__(int i)
                                      {
                                          double x = d_x_train_ptr[i * D + split.cut_feature];
                                          return x <= split.cut_value;
                                      });

        thrust::stable_partition_copy(thrust::device,
                                      thrust::make_counting_iterator(0), thrust::make_counting_iterator(N * D),
                                      left_x_indices.begin(), right_x_indices.begin(),
                                      [=] __device__(int i)
                                      {
                                          double x = d_x_train_ptr[(i / D) * D + split.cut_feature];
                                          return x <= split.cut_value;
                                      });

        thrust::device_vector<double> left_x_train(left_indices.size() * D);
        thrust::device_vector<double> right_x_train(right_indices.size() * D);
        thrust::device_vector<double> left_y_train(left_indices.size());
        thrust::device_vector<double> right_y_train(right_indices.size());

        thrust::gather(thrust::device, left_indices.begin(), left_indices.end(), d_y_train.begin(), left_y_train.begin());
        thrust::gather(thrust::device, right_indices.begin(), right_indices.end(), d_y_train.begin(), right_y_train.begin());
        thrust::gather(thrust::device, left_x_indices.begin(), left_x_indices.end(), d_x_train.begin(), left_x_train.begin());
        thrust::gather(thrust::device, right_x_indices.begin(), right_x_indices.end(), d_x_train.begin(), right_x_train.begin());

        // recursively build left and right subtrees
        tree_node_t *left = build_cart_helper(D, left_y_train.size(), left_x_train, left_y_train, depth - 1);
        tree_node_t *right = build_cart_helper(D, right_y_train.size(), right_x_train, right_y_train, depth - 1);

        tree_node_t *node = (tree_node_t *)malloc(sizeof(tree_node_t));
        node->cut_feature = split.cut_feature;
        node->cut_value = split.cut_value;
        node->left = left;
        node->right = right;
        node->prediction = mean;
        left->parent = node;
        right->parent = node;
        return node;
    }
}

// IMPLEMENTATION

void init(int D, int train_size, int test_size, std::vector<double> &x_train, std::vector<double> &y_train, std::vector<double> &x_test, std::vector<double> &y_test)
{
    D_global = D;
    train_size_global = train_size;
    test_size_global = test_size;
    x_train_global = new thrust::device_vector<double>(x_train.begin(), x_train.end());
    y_train_global = new thrust::device_vector<double>(y_train.begin(), y_train.end());
    x_test_global = x_test;
    y_test_global = y_test;
    predictions_global = std::vector<double>(test_size_global);
}

tree_node_t *build_cart(int depth)
{
    return build_cart_helper(D_global, train_size_global, *x_train_global, *y_train_global, depth);
}

tree_node_t *build_cart_iterative(int depth)
{
    std::vector<tree_node_t *> tree;
    using XYpair = std::pair<thrust::device_vector<double>, thrust::device_vector<double>>;

    // Pair of (x training data, y training data)
    std::vector<XYpair> split_data_curr(1);
    std::vector<XYpair> split_data_temp;
    split_data_curr[0] = XYpair(*x_train_global, *y_train_global);

    omp_set_num_threads(NUM_THREADS);

    // Array of CUDA Streams
    hipStream_t streams[NUM_STREAMS];

    for (int stream_idx = 0; stream_idx < NUM_STREAMS; ++stream_idx)
    {
        hipStreamCreate(&streams[stream_idx]);
    }

    int current_layer_idx = 0;
    for (int i = 0; i <= depth; ++i)
    {
        int curr_level_size = pow(2, i);
        std::vector<bool> threadFinished(curr_level_size, true);
        std::vector<tree_node_t *> curr_level(curr_level_size);

        split_data_temp.resize(pow(2, i + 1));

#pragma omp parallel for
        for (int j = 0; j < curr_level_size; ++j)
        {
            int current_idx = current_layer_idx + j;
            thrust::device_vector<double> x_train_curr;
            thrust::device_vector<double> y_train_curr;

            x_train_curr = split_data_curr[j].first;
            y_train_curr = split_data_curr[j].second;

            int N = y_train_curr.size();

            double weight = 1.0 / N;
            double mean = thrust::reduce(thrust::cuda::par.on(streams[j % NUM_STREAMS]), y_train_curr.begin(), y_train_curr.end(), 0.0) / N;

            int parent_idx = (current_idx - 1) / 2;
            bool isLeftChild = (current_idx - 1) % 2 == 0;

            // Case where there should be no node at this index
            // If the parent node is a leaf or the parent node is also null, we push back null
            if (current_idx != 0 && (tree[parent_idx] == NULL || tree[parent_idx]->cut_feature == -1))
            {
                curr_level[j] = NULL;
                continue;
            }
            // Else if the current node's parent is not a leaf but the stopping criteria is reached, push back leaf
            else if (i == depth || elements_equal_gpu(thrust::cuda::par.on(streams[j % NUM_STREAMS]), y_train_curr, N, THRESHOLD) ||
                     rows_equal_gpu(thrust::cuda::par.on(streams[j % NUM_STREAMS]), x_train_curr, D_global, N, THRESHOLD))
            {
                tree_node_t *leaf = (tree_node_t *)malloc(sizeof(tree_node_t));
                leaf->left = NULL;
                leaf->right = NULL;
                leaf->parent = NULL;
                leaf->prediction = mean;
                leaf->cut_feature = -1;
                leaf->cut_value = NAN;
                curr_level[j] = leaf;

                if (tree[parent_idx]->cut_feature != -1)
                {
                    if (isLeftChild)
                    {
                        tree[parent_idx]->left = leaf;
                    }
                    else
                    {
                        tree[parent_idx]->right = leaf;
                    }
                }

                continue;
            }

            threadFinished[j] = false;

            split_output_t split = split_gpu(thrust::cuda::par.on(streams[j % NUM_STREAMS]), D_global, N, x_train_curr, y_train_curr);

            thrust::device_ptr<double> x_train_curr_ptr = x_train_curr.data();
            int D_local = D_global;

            int size_left = thrust::count_if(thrust::cuda::par.on(streams[j % NUM_STREAMS]),
                                             thrust::make_counting_iterator(0), thrust::make_counting_iterator(N),
                                             [x_train_curr_ptr, split, D_local] __device__(int i)
                                             {
                                                 double x = x_train_curr_ptr[i * D_local + split.cut_feature];
                                                 return x <= split.cut_value;
                                             });
            int size_right = N - size_left;

            thrust::device_vector<int> left_indices(size_left);
            thrust::device_vector<int> right_indices(size_right);
            thrust::device_vector<int> left_x_indices(size_left * D_global);
            thrust::device_vector<int> right_x_indices(size_right * D_global);

            thrust::stable_partition_copy(thrust::cuda::par.on(streams[j % NUM_STREAMS]),
                                          thrust::make_counting_iterator(0), thrust::make_counting_iterator(N),
                                          left_indices.begin(), right_indices.begin(),
                                          [x_train_curr_ptr, split, D_local] __device__(int i)
                                          {
                                              double x = x_train_curr_ptr[i * D_local + split.cut_feature];
                                              return x <= split.cut_value;
                                          });

            thrust::stable_partition_copy(thrust::cuda::par.on(streams[j % NUM_STREAMS]),
                                          thrust::make_counting_iterator(0), thrust::make_counting_iterator(N * D_global),
                                          left_x_indices.begin(), right_x_indices.begin(),
                                          [x_train_curr_ptr, split, D_local] __device__(int i)
                                          {
                                              double x = x_train_curr_ptr[(i / D_local) * D_local + split.cut_feature];
                                              return x <= split.cut_value;
                                          });

            thrust::device_vector<double> left_x_train(left_indices.size() * D_global);
            thrust::device_vector<double> right_x_train(right_indices.size() * D_global);
            thrust::device_vector<double> left_y_train(left_indices.size());
            thrust::device_vector<double> right_y_train(right_indices.size());

            thrust::gather(thrust::cuda::par.on(streams[j % NUM_STREAMS]), left_indices.begin(), left_indices.end(), y_train_curr.begin(), left_y_train.begin());
            thrust::gather(thrust::cuda::par.on(streams[j % NUM_STREAMS]), right_indices.begin(), right_indices.end(), y_train_curr.begin(), right_y_train.begin());
            thrust::gather(thrust::cuda::par.on(streams[j % NUM_STREAMS]), left_x_indices.begin(), left_x_indices.end(), x_train_curr.begin(), left_x_train.begin());
            thrust::gather(thrust::cuda::par.on(streams[j % NUM_STREAMS]), right_x_indices.begin(), right_x_indices.end(), x_train_curr.begin(), right_x_train.begin());

            // Insert data for the next level
            split_data_temp[j * 2] = XYpair(left_x_train, left_y_train);
            split_data_temp[j * 2 + 1] = XYpair(right_x_train, right_y_train);

            tree_node_t *node = (tree_node_t *)malloc(sizeof(tree_node_t));
            node->cut_feature = split.cut_feature;
            node->cut_value = split.cut_value;
            node->left = NULL;
            node->right = NULL;
            node->prediction = mean;

            if (current_idx != 0)
            {
                if (isLeftChild)
                {
                    tree[parent_idx]->left = node;
                }
                else
                {
                    tree[parent_idx]->right = node;
                }
                node->parent = tree[parent_idx];
            }
            curr_level[j] = node;
        }

        // update the starting index of the next layer
        current_layer_idx += curr_level_size;

        tree.insert(tree.end(), curr_level.begin(), curr_level.end());

        // Check if the entire layer is NULL or Leaves
        bool allTrue = std::all_of(threadFinished.begin(), threadFinished.end(), [](bool element)
                                   { return element; });
        if (allTrue)
        {
            break;
        }

        // populate the split data for the next layer
        split_data_curr = split_data_temp;
        split_data_temp.clear();
    }
    return tree[0];
}

/** Recursive helper for evaluating an input data point using a tree */
double eval_helper_gpu(tree_node_t *tree, std::vector<double> &data)
{
    if (tree->left == NULL && tree->right == NULL)
    {
        return tree->prediction;
    }

    int feature = tree->cut_feature;
    double cut_value = tree->cut_value;

    if (data[feature] <= cut_value)
    {
        return eval_helper_gpu(tree->left, data);
    }
    else
    {
        return eval_helper_gpu(tree->right, data);
    }
}

void compute_predictions(tree_node_t *tree)
{
    for (int i = 0; i < test_size_global; ++i)
    {
        std::vector<double> data = std::vector<double>(x_test_global.begin() + i * D_global, x_test_global.begin() + i * D_global + D_global);
        double prediction = eval_helper_gpu(tree, data);
        predictions_global[i] = prediction;
    }
}

double eval_mse(tree_node_t *tree)
{
    compute_predictions(tree);
    // compute MSE
    double accumulator = 0;
    for (int i = 0; i < test_size_global; ++i)
    {
        accumulator += pow(predictions_global[i] - y_test_global[i], 2);
    }

    return accumulator / test_size_global;
}

double eval_classification(tree_node_t *tree)
{
    compute_predictions(tree);

    // compute classification error
    double accumulator = 0;
    for (int i = 0; i < test_size_global; ++i)
    {
        if (predictions_global[i] != y_test_global[i])
        {
            accumulator += 1;
        }
    }

    return accumulator / test_size_global;
}

struct RandomGenerator
{
    int a, b;

    __host__ __device__
    RandomGenerator(int _a, int _b) : a(_a), b(_b) {};

    __host__ __device__ float operator()(const unsigned int n) const
    {
        thrust::default_random_engine rng;
        thrust::uniform_int_distribution<int> dist(a, b);
        rng.discard(n);

        return dist(rng);
    }
};

forest_t *build_forest(int depth, int num_trees)
{
    int subsample_size = ceil(SUBSAMPLE_RATE * train_size_global);
    forest_t *trees = new std::vector<tree_node_t *>(num_trees);
    thrust::device_vector<double> x_train_rand(subsample_size * D_global);
    thrust::device_vector<double> y_train_rand(subsample_size);

    thrust::device_vector<int> random_idx(subsample_size);
    tree_node_t *curr_tree;
    int D_local = D_global;

    thrust::device_ptr<double> x_train_global_ptr = (*x_train_global).data();
    thrust::device_ptr<double> x_train_rand_ptr = x_train_rand.data();
    thrust::device_ptr<int> random_idx_ptr = random_idx.data();

    for (int i = 0; i < num_trees; i++)
    {
        thrust::counting_iterator<unsigned int> index_sequence_begin(i * subsample_size);
        thrust::transform(thrust::device, index_sequence_begin,
                          index_sequence_begin + random_idx.size(),
                          random_idx.begin(),
                          RandomGenerator(0, train_size_global - 1));

        thrust::gather(thrust::device, random_idx.begin(), random_idx.end(),
                       (*y_train_global).begin(), y_train_rand.begin());

        for (int d = 0; d < D_global; d++)
        {
            thrust::counting_iterator<unsigned int> counting_iter(0);
            thrust::for_each(thrust::device,
                             counting_iter,
                             counting_iter + subsample_size,
                             [=] __device__(int idx)
                             {
                                 x_train_rand_ptr[idx * D_local + d] = x_train_global_ptr[random_idx_ptr[idx] * D_local + d];
                             });
        }
        curr_tree = build_cart_helper(D_global, subsample_size, x_train_rand, y_train_rand, depth);
        (*trees)[i] = curr_tree;
    }

    return trees;
}

double eval_forest_mse(forest_t *forest)
{
    double weight = 1.0 / (*forest).size();

    for (int i = 0; i < (*forest).size(); i++)
    {
        for (int j = 0; j < test_size_global; j++)
        {
            std::vector<double> data = std::vector<double>(x_test_global.begin() + j * D_global, x_test_global.begin() + j * D_global + D_global);
            double prediction = eval_helper_gpu((*forest)[i], data);
            predictions_global[j] += weight * prediction;
        }
    }

    // compute MSE
    double accumulator = 0;
    for (int i = 0; i < test_size_global; ++i)
    {
        accumulator += pow(predictions_global[i] - y_test_global[i], 2);
    }

    return accumulator / test_size_global;
}
