#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <chrono>
#include <cmath>
#include <cstring>
#include <fstream>
#include <iostream>
#include <random>
#include <string>
#include <vector>

#include "common.h"

// =================
// Helper Functions
// =================

// Command Line Option Processing
int find_arg_idx(int argc, char **argv, const char *option) {
  for (int i = 1; i < argc; ++i) {
    if (strcmp(argv[i], option) == 0) {
      return i;
    }
  }
  return -1;
}

char *find_string_option(int argc, char **argv, const char *option,
                         char *default_value) {
  int iplace = find_arg_idx(argc, argv, option);

  if (iplace >= 0 && iplace < argc - 1) {
    return argv[iplace + 1];
  }

  return default_value;
}

// ==============
// Main Function
// ==============

int main(int argc, char **argv) {
  // Parse Args
  if (find_arg_idx(argc, argv, "-h") >= 0) {
    std::cout << "Options:" << std::endl;
    std::cout << "-h: see this help" << std::endl;
    std::cout << "-f: dataset csv file name" << std::endl;
    return 0;
  }

  char *dataset_file_name =
      find_string_option(argc, argv, "-f", "Admission_Predict.csv");

  std::ifstream dataset_file;
  dataset_file.open("datasets/" + dataset_file_name);
  if (!dataset_file.is_open()) {
    std::cerr << "Error opening dataset" << std::endl;
    return -1;
  }

  std::string line;
  // get past the header line
  std::getline(dataset_file, line);
  std::vector<std::vector<double>> csvData;
  while (std::getline(dataset_file, line)) {
    std::stringstream ss(line);
    std::vector<std::string> row;
    std::string cell;
    while (std::getline(ss, cell, ',')) {
      row.push_back((double)cell);
    }
    csvData.push_back(row);
  }
  dataset_file.close();

  int N = csvData.size();
  // last dimension is the y_value
  int D = csvData[0].size() - 1;
  int split = ceil(0.8 * N);
  double *x_train = malloc(split * D * sizeof(double));
  double *x_test = malloc((N - split) * D * sizeof(double));
  double *y_train = malloc(split * sizeof(double));
  double *y_test = malloc((N - split) * sizeof(double));
  double *accuracy = malloc(sizeof(double));

  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < D; ++j) {
      if (i < split) {
        x_train[i * D + j] = csvData[i][j];
      } else {
        x_test[i * D + j] = csvData[i][j];
      }
    }
    if (i < split) {
      y_train[i] = csvData[i][D];
    } else {
      y_test[i] = csvData[i][D];
    }
  }

  std::cout << "Size of x train: " << sizeof(x_train) / sizeof(x_train[0])
            << std::endl;

  // copy data to the gpu
  double *x_train_gpu;
  double *y_train_gpu;
  double *x_test_gpu;
  double *y_test_gpu;
  double *accuracy_gpu;
  hipMalloc((void **)&x_train_gpu, split * D * sizeof(double));
  hipMalloc((void **)&y_train_gpu, split * sizeof(double));
  hipMalloc((void **)&x_test_gpu, (N - split) * D * sizeof(double));
  hipMalloc((void **)&y_test_gpu, (N - split) * sizeof(double));
  hipMalloc((void **)&accuracy_gpu, 1 * sizeof(double));

  hipMemcpy(x_train_gpu, x_train, split * D * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(x_test_gpu, x_test, (N - split) * D * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(y_train_gpu, y_train, split * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(y_test_gpu, y_test, (N - split) * sizeof(double),
             hipMemcpyHostToDevice);

  // track start time
  auto start_time = std::chrono::steady_clock::now();

  // train and predict

  hipDeviceSynchronize();
  auto end_time = std::chrono::steady_clock::now();

  std::chrono::duration<double> diff = end_time - start_time;
  double seconds = diff.count();

  hipMemcpy(accuracy, accuracy_gpu, sizeof(double), hipMemcpyDeviceToHost);

  // Finalize
  std::cout << "Training and Prediction Time = " << seconds << " seconds for "
            << num_parts << " particles." << std::endl;
  std::cout << "Accuracy = " << *accuracy << std::endl;
  hipFree(x_train_gpu);
  hipFree(x_test_gpu);
  hipFree(y_train_gpu);
  hipFree(y_test_gpu);
  free(x_train);
  free(x_test);
  free(y_train);
  free(y_test);
}
