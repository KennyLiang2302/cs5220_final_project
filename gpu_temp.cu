#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>


int round_down(int x, int D) {
    return x - (x % D);
} 

thrust::device_vector<int> argsort_exhaustive(thrust::device_vector<double>& x, int d, int D, int N) {
    thrust::device_vector<int> indices(D * N);
    thrust::sequence(indices.begin(), indices.end());

    thrust::sort(indices.begin(), indices.end(), [x] __device__ (int left_idx, int right_idx) {
        return x[round_down(left_idx, D) + d] < x[round_down(right_idx, D) + d];
    })
    return indices;
}

thrust::device_vector<int> argsort(thrust::device_vector<double>& x, int d, int D, int N) {
    thrust::device_vector<int> indices(N);
    thrust::sequence(indices.begin(), indices.end());

    thrust::sort(indices.begin(), indices.end(), [x] __device__ (int left_idx, int right_idx) {
        return x[left_idx*D + d] < x[right_idx*D + d];
    })
    return indices;
}

split_output_t split_serial(int D, int N, thrust::device_vector<double> &x_train, thrust::device_vector<double> &y_train)
{
    double weight = 1.0 / N;
    double min_loss = std::numeric_limits<double>::infinity();
    int feature = std::numeric_limits<int>::infinity();
    double cut_value = std::numeric_limits<double>::infinity();

    thrust::vector<split_output_t> split_results ((N-1) * D);

    // iterate through each feature
    for (int d = 0; d < D; ++d)
    {
        thrust::device_vector<int> indices_x = argsort_exhaustive(x_train, d, D, N);
        thrust::device_vector<int> indices_y = argsort(x_train, d, D, N);

        thrust::device_vector<double> x_train_sorted(N*D);
        thrust::device_vector<double> y_train_sorted(N);

        thrust::gather(thrust::device, indices_x.begin(), indices_x.end(), x_train.begin(), x_train_sorted.begin());
        thrust::gather(thrust::device, indices_y.begin(), indices_y.end(), y_train.begin(), y_train_sorted.begin());
        thrust::vector<double> y_train_sorted_squared(N);
        thrust::transform(y_train_sorted.begin(), y_train_sorted.end(), y_train_sorted_squared.begin(),
                       [] __device__ (double x) -> double
                       { return x * x; });

        double mean_square_left = 0.0;
        double mean_left = 0.0;
        double weight_left = 0.0;
        double mean_square_right = weight * thrust::reduce(y_train_sorted_squared.begin(), y_train_sorted_squared.end(), 0.0);
        double mean_right = weight * thrust::reduce(y_train_sorted.begin(), y_train_sorted.end(), 0.0);
        double weight_right = 1.0;

        thrust::device_vector<double> y_prefix_sum(N);
        thrust::device_vector<double> y_squared_prefix_sum(N);

        thrust::inclusive_scan(thrust::host, y_train_sorted.begin(), y_train_sorted.end(), y_prefix_sum.begin());
        thrust::inclusive_scan(thrust::host, y_train_sorted_squared.begin(), y_train_sorted_squared.begin(), y_squared_prefix_sum.begin());

        thrust::device_vector<int> split_indices(N-1);
        thrust::sequence(split_indices.begin(), split_indices.end());

        thrust::transform(split_indices.begin(), split_indices.end(), losses.begin(), 
                [N, d, split_results, weight, mean_square_right, mean_right, y_prefix_sum, y_squared_prefix_sum, x_train_sorted] __device__ (int index) {
                double mean_square_left = weight * y_squared_prefix_sum[index];
                double mean_left = weight * y_prefix_sum[index];
                double local_mean_right = mean_right - mean_left; 
                double local_mean_sq_right = mean_square_right - mean_square_left;
                double weight_left = (index + 1) * weight;
                double weight_right = (N - index - 1) * weight;
                double left_loss = mean_square_left - pow(mean_left, 2) / weight_left;
                double right_loss = local_mean_sq_right - pow(local_mean_right, 2) / weight_right;
                split_output_t split_output;
                split_output.cut_feature = d;
                split_output.cut_value = (x_train_sorted[index] + x_train_sorted[index + 1]) / 2;
                split_output.loss = left_loss + right_loss;
                split_results[(N-1)*d + index] = split_output;
            });

        hipDeviceSynchronize();
    }

    split_output_t output = thrust::reduce(thrust::device, split_results.begin(), split_results.end(), split_results[0], 
            [] __device__ (split_output_t& left, split_output_t& right) {
            if (left.loss < right.loss) {
                return left;
            } else {
                return right;
            }
        });

    return output;
}
